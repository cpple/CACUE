#include "hip/hip_runtime.h"
/*
 Copyright (c) 2016, David lu
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions are met:
 * Redistributions of source code must retain the above copyright
 notice, this list of conditions and the following disclaimer.
 * Redistributions in binary form must reproduce the above copyright
 notice, this list of conditions and the following disclaimer in the
 documentation and/or other materials provided with the distribution.
 * Neither the name of the <organization> nor the
 names of its contributors may be used to endorse or promote products
 derived from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
 EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY
 DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda_log.h"
#include "../math_utils.h"


__global__ void _k_CACU_SUMBYSIZE_BYWIDTH_GPU(float_t *x, int heigth, int width, float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	float_t *start;

	for (int i = bid; i < heigth; i += BLOCKNUM) {
		start = x + i * width;
		for(int j = tid ;  j < width; j += THREADNUM)
			y[i] += start[j];
	}
}

__global__ void _k_CACU_SUMBYSIZE_BYHEIGHT_GPU(float_t *x, int height, int width, float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	float_t *start;

	for (int i = bid; i < width; i += BLOCKNUM) {
		start = x + i;
		for(int j = tid ;j < height; j += THREADNUM)
			y[i] += start[j*width];
	}
}

extern "C" void cacu_sumbysize_gpu(SUM SUMTYPE ,float_t *x, int length, float_t *y, int width){

	int height = length / width;

	if (BYWIDTH == SUMTYPE)
		_k_CACU_SUMBYSIZE_BYWIDTH_GPU<<<BLOCKNUM, THREADNUM, 0>>>(x, height,width, y);
	else if(BYHEIGHT == SUMTYPE)
		_k_CACU_SUMBYSIZE_BYHEIGHT_GPU<<<BLOCKNUM, THREADNUM, 0>>>(x, height,width, y);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_CXSIZE_GPU(float_t *x, int length, float_t *a, int size,float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	int block_size = length / size;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		y[i] = x[i]*a[i / block_size];
	}
}

extern "C" void cacu_cxsize_gpu(float_t *x, int length, float_t *a, int size,float_t *y)
{
	_k_CACU_CXSIZE_GPU<<<BLOCKNUM, THREADNUM, 0>>>(x, length, a, size, y);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_SXSIZE_GPU(float_t *x, int length, float_t a,float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		y[i] = x[i]*a;
	}
}

extern "C" void cacu_sxsize_gpu(float_t *x, int length, float_t a, float_t *y)
{
	_k_CACU_SXSIZE_GPU<<<BLOCKNUM, THREADNUM, 0>>>(x, length, a, y);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_CDXSIZE_GPU(float_t *x, int length, float_t *a, int size ,float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	int block_size = length / size;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		y[i] = x[i]/a[i / block_size];
	}
}

extern "C" void cacu_cdxsize_gpu(float_t *x, int length, float_t *a, int size, float_t *y)
{
	_k_CACU_CDXSIZE_GPU<<<BLOCKNUM, THREADNUM, 0>>>(x, length, a, size, y);
	CUDA_CHECK(hipDeviceSynchronize());
}

extern "C" void cacu_sdxsize_gpu(float_t *x, int length, float_t a, float_t *y);

__global__ void _k_CACU_SSXPY_GPU(float_t *x, float_t a, int size, float_t *y, float_t b, int length, float_t *z) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	int block_size = length / size;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		z[i] = b*y[i] + a*x[i / block_size];
	}
}


extern "C" void cacu_ssxpy_gpu(float_t *x, float_t a, int size, float_t *y, float_t b, int length, float_t *z)
{
	_k_CACU_SSXPY_GPU<<<BLOCKNUM, THREADNUM, 0>>>(x, a, size, y, b, length, z);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_SQR_GPU(float_t *x, int length, float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		y[i] = x[i] * x[i];
	}
}

extern "C" void cacu_sqr_gpu(float_t *x, int length, float_t *y)
{
	_k_CACU_SQR_GPU<<<BLOCKNUM, THREADNUM, 0>>>(x, length, y);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_ROOT_GPU(float_t *x, int length, float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		y[i] = sqrt(x[i]);
	}
}

extern "C" void cacu_root_gpu(float_t *x, int length, float_t *y)
{
	_k_CACU_ROOT_GPU<<<BLOCKNUM, THREADNUM, 0>>>(x, length, y);
	CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void _k_CACU_STDBYCHANNEL_GPU(float_t *varience, int length, float_t *std, float_t epsilon) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		std[i] = sqrt(varience[i] + epsilon);
	}
}

extern "C" void cacu_stdbychannel_gpu(float_t *varience, int length, float_t *std, float_t epsilon)
{
	_k_CACU_STDBYCHANNEL_GPU<<<BLOCKNUM, THREADNUM, 0>>>(varience, length, std, epsilon);
	CUDA_CHECK(hipDeviceSynchronize());
}


__global__ void _k_CACU_SSX_GPU(float_t *x, int length, float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		y[i] *= x[i];
	}
}

extern "C" void cacu_ssx_gpu(float_t *x, int length, float_t *y)
{
	_k_CACU_SSX_GPU<<<BLOCKNUM, THREADNUM, 0>>>(x, length, y);
	CUDA_CHECK(hipDeviceSynchronize());
}
