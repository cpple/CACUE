#include "hip/hip_runtime.h"
/*
 Copyright (c) 2016, David lu
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions are met:
 * Redistributions of source code must retain the above copyright
 notice, this list of conditions and the following disclaimer.
 * Redistributions in binary form must reproduce the above copyright
 notice, this list of conditions and the following disclaimer in the
 documentation and/or other materials provided with the distribution.
 * Neither the name of the <organization> nor the
 names of its contributors may be used to endorse or promote products
 derived from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
 EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY
 DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda_log.h"



__global__ void _k_CACU_ISAXB_GPU(float_t *x, int length, float_t a ,unsigned int *index_, float_t b, float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {

		if(*index_ != i)
			y[i] = x[i];
		else
			y[i] = a*x[i] + b;

	}
}

/**
 * @cacu_isaxdb_gpu
 * y[index] = x[index]*a + b
 */
extern "C" void cacu_isaxb_gpu(float_t *x, int length, float_t a ,unsigned int *index_, float_t b, float_t *y) {

	_k_CACU_ISAXB_GPU<<<BLOCKNUM, THREADNUM, 0>>>(x, length,a ,index_,b,y);

	CUDA_CHECK(hipDeviceSynchronize());

}
