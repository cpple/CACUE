#include "hip/hip_runtime.h"
/*
Copyright (c) 2016, David lu
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
* Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.
* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.
* Neither the name of the <organization> nor the
names of its contributors may be used to endorse or promote products
derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "cuda_log.h"


/*
 *channel: channel of input data
 *kernel_size: pooling window size
 *input_dim: width of input data
 *output_dim: width of output data
 */
__global__ void _k_CACU_CROSS_ENTROPY_GPU(float_t *x, unsigned int *label_, float_t loss_) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	if(threadid == 0)
		loss_ -= log(x[*label_]);
}


extern "C" void cacu_cross_entropy_gpu(float_t *x, unsigned int *label_, float_t loss_){

	_k_CACU_CROSS_ENTROPY_GPU<<<1, 1, 0>>>(x, label_,loss_);
	CUDA_CHECK(hipDeviceSynchronize());
}




