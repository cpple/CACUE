#include "hip/hip_runtime.h"
/*
 Copyright (c) 2016, David lu
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions are met:
 * Redistributions of source code must retain the above copyright
 notice, this list of conditions and the following disclaimer.
 * Redistributions in binary form must reproduce the above copyright
 notice, this list of conditions and the following disclaimer in the
 documentation and/or other materials provided with the distribution.
 * Neither the name of the <organization> nor the
 names of its contributors may be used to endorse or promote products
 derived from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY
 EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY
 DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "cuda_log.h"

#include "../../utils/data_defination.h"

__global__ void _k_CACU_SAXPY_ATOMIC_GPU(const float_t *x, float_t a, float_t *y,int length){

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {

		atomicAdd(y + i, a * x[i]);
	}

}

extern "C" void cacu_saxpy_atomic_gpu(const float_t *x, float_t a, float_t *y,int length)
{
	_k_CACU_SAXPY_ATOMIC_GPU<<<BLOCKNUM, THREADNUM, 0>>>(x, a ,y,length);

	CUDA_CHECK(hipDeviceSynchronize());
}


__global__ void _k_CACU_ISAXB_GPU(const mycnn::float_t *x, int length,const mycnn::float_t a ,const unsigned int *index_, const mycnn::float_t b, mycnn::float_t *y) {

	int tid = threadIdx.x;
	int bid = blockIdx.x;

	int threadid = bid * THREADNUM + tid;

	for (int i = threadid; i < length; i += BLOCKNUM * THREADNUM) {
		y[i] = x[i];
	}

	__syncthreads();

	if(threadid == 0)
		y[index_[0]] = a*x[index_[0]] + b;
}

/**
 * @cacu_isaxdb_gpu
 * y[index] = x[index]*a + b
 */
extern "C" void cacu_isaxb_gpu(const mycnn::float_t *x, int length,const mycnn::float_t a ,const unsigned int *index_,const mycnn::float_t b, mycnn::float_t *y) {

	_k_CACU_ISAXB_GPU<<<BLOCKNUM, THREADNUM, 0>>>(x, length,a ,index_,b,y);

	CUDA_CHECK(hipDeviceSynchronize());

}

__global__ void _k_ARGMAX_GPU(const mycnn::float_t *x, int length, unsigned int *index_) {

	__shared__ mycnn::float_t shared_data[THREADNUM];

	__shared__ unsigned int index_data[THREADNUM];

	int tid = threadIdx.x;

	int max_length = THREADNUM;
	if(THREADNUM > length)
		max_length = length;

	if(tid < max_length){
		shared_data[tid] = x[tid];
		index_data[tid] = tid;
	}

	for (unsigned int i = tid; i < length; i += THREADNUM) {
		if(x[i] > shared_data[tid])
		{
			shared_data[tid] = x[i];
			index_data[tid] = i;
		}
	}

	__syncthreads();

	if(tid == 0){

		for(int i = 1; i< max_length; ++i)
		{
			if(shared_data[0] < shared_data[i]){
				shared_data[0] = shared_data[i];
				index_data[0] = index_data[i];
			}
		}
		index_[0] = index_data[0];
	}
}

extern "C" void cacu_argmax_gpu(const mycnn::float_t *x,int length, unsigned int *index_)
{
	_k_ARGMAX_GPU<<<1, THREADNUM, 0>>>(x, length, index_);

	CUDA_CHECK(hipDeviceSynchronize());
}

extern "C" void cacu_transpose_gpu(mycnn::float_t *mtx, const int m, const int n)
{

}
